/******************************************************************************
 *cr
 *cr            (C) Copyright 2010 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ******************************************************************************/

#include <stdio.h>
#include <iostream>
#include "support.h"
#include "kernel.cu"
#include "kernel2.cu"
#include "kernel3.cu"
#include "kernel4.cu"

int main(int argc, char* argv[])
{
	Timer timer;
	if (argc != 3) {
	    printf("\nInput files not specified");
	    exit(0);
	}
        char *inputImageFile = argv[1];
        char *labelFile = argv[2];

	// Initialize host variables ----------------------------------------------
	printf("\nSetting up the problem..."); fflush(stdout);
	startTime(&timer);
        
	// Allocate and initialize host variables ----------------------------------------------
	Matrix *conv_weight, conv_bias, fc1_weight, fc1_bias, fc2_weight, fc2_bias, test_image;
 
        input(inputImageFile, &conv_weight, &conv_bias, &fc1_weight, &fc1_bias, &fc2_weight, &fc2_bias, &test_image);

	dim3 dim_grid(1,8) , dim_block(28,28);
 	dim3 dim_grid2(512,8) , dim_block2(28,28);
  dim3 dim_grid3(1,1), dim_block3(16,32);
  dim3 dim_grid4(1,1), dim_block4(10,1);
 
	int result,*results;

	stopTime(&timer); 
	printf("%f s\n", elapsedTime(timer));

	// Allocate device variables ----------------------------------------------

	//INSERT DEVICE ALLOCATION CODE HERE
	
  printf("Allocating device variables..."); fflush(stdout);
	startTime(&timer);
  Matrix fc1_weight_d, fc2_weight_d, test_image_d;
  //Matrix A;
  Matrix conv_weight_1;
  Matrix padded_image=allocateMatrix(37,37);

  
hipMalloc(&conv_weight_1.elements,(8*10*10+30)*sizeof(float));
hipMalloc(&test_image_d.elements,28*28*sizeof(float));
//hipMalloc(&(A).elements,8*28*28*sizeof(float));
hipMalloc(&fc1_weight_d.elements,8*28*28*512*sizeof(float));
hipMalloc(&fc2_weight_d.elements,10*512*sizeof(float));
hipMalloc(&results,sizeof(int));


	hipDeviceSynchronize();
	stopTime(&timer); 
	printf("%f s\n", elapsedTime(timer));
 //padding
 for(int i=0;i<37;i++)for(int j=0;j<37;j++) padded_image.elements[i*28+j]=0;
 for(int i=4;i<(32);i++){
   for(int j=4;j<(32);j++){
padded_image.elements[i*37+j]=test_image.elements[(i-4)*28+(j-4)];
}}
//*/ 
 
	// Copy host variables to device ------------------------------------------

	printf("Copying data from host to device..."); fflush(stdout);
	startTime(&timer);

	//INSERT HOST TO DEVICE COPY CODE HERE
hipError_t cuda_ret;

cuda_ret=hipMemcpyToSymbol(HIP_SYMBOL(padded_img),padded_image.elements,37*37*sizeof(float));
if(cuda_ret!=hipSuccess)FATAL("unable to allocate padded_img");


cuda_ret=hipMemcpyToSymbol(HIP_SYMBOL(conv_ww),conv_weight[0].elements,830*sizeof(float));
if(cuda_ret!=hipSuccess)FATAL("unable to allocate conv_ww");

cuda_ret=hipMemcpyToSymbol(HIP_SYMBOL(conv_b),conv_bias.elements,8*sizeof(float));
if(cuda_ret!=hipSuccess)FATAL("unable to allocate conv_bias");

cuda_ret=hipMemcpy(fc1_weight_d.elements,fc1_weight.elements,8*28*28*512*sizeof(float),hipMemcpyHostToDevice);
if(cuda_ret!=hipSuccess)FATAL("unable to allocate fc1_weight_d");

cuda_ret=hipMemcpy(fc2_weight_d.elements,fc2_weight.elements,10*512*sizeof(float),hipMemcpyHostToDevice);
if(cuda_ret!=hipSuccess)FATAL("unable to allocate fc2_weight_d");

cuda_ret=hipMemcpy(results,&result,sizeof(int),hipMemcpyHostToDevice);
if(cuda_ret!=hipSuccess)FATAL("unable to allocate results");


cuda_ret=hipMemcpyToSymbol(HIP_SYMBOL(fc1_b),fc1_bias.elements,512*sizeof(float));
if(cuda_ret!=hipSuccess)FATAL("unable to allocate fc1_bias");

cuda_ret=hipMemcpyToSymbol(HIP_SYMBOL(fc2_b),fc2_bias.elements,10*sizeof(float));
if(cuda_ret!=hipSuccess)FATAL("unable to allocate fc2_bias");



	hipDeviceSynchronize();
	stopTime(&timer); 
	printf("%f s\n", elapsedTime(timer));

	// Launch kernel ----------------------------------------------------------
	printf("Launching kernel..."); fflush(stdout);
	startTime(&timer);

	//INSERT KERNEL LAUNCH CODE HERE
//---------
//void convolution(float *conv_w, float *image,float*fc1_w, float* fc2_w){


kernel1<<<dim_grid,dim_block>>>();
kernel2<<<dim_grid2,dim_block2>>>(fc1_weight_d.elements);
kernel3<<<dim_grid3,dim_block3>>>(fc1_weight_d.elements,fc2_weight_d.elements);
kernel4<<<dim_grid3,dim_block3>>>(results);


//---------


	cuda_ret = hipDeviceSynchronize();
	if(cuda_ret != hipSuccess) FATAL("Unable to launch/execute kernel");

	hipDeviceSynchronize();
	stopTime(&timer); 
	printf("%f s\n", elapsedTime(timer));

	// Copy device variables from host ----------------------------------------

	printf("Copying data from device to host..."); fflush(stdout);
	startTime(&timer);

	//INSERT DEVICE TO HOST COPY CODE HERE
cuda_ret=hipMemcpy(&result,results,sizeof(int),hipMemcpyDeviceToHost);
if(cuda_ret!=hipSuccess)FATAL("unable to allocate result");

	hipDeviceSynchronize();
	stopTime(&timer); 
	printf("%f s\n", elapsedTime(timer));
printf("\n\nRESULT is ......%d\n\n",result);

	// Verify correctness -----------------------------------------------------
        verify(result, labelFile);

	// Free host and device memory ------------------------------------------------------------

	return 0;
}


