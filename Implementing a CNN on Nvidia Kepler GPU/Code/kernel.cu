
#include <hip/hip_runtime.h>
/******************************************************************************
 *cr
 *cr            (C) Copyright 2010 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ******************************************************************************/


//INSERT KERNEL CODE HERE
__constant__ float conv_b[8];
__constant__ float fc1_b[512];
__constant__ float fc2_b[10];
__constant__ float conv_ww[830];
__constant__ float padded_img[37*37];

__device__ float reshape[28*28*8];
//__device__ float L[10];
__device__ float I[512];


__global__ void kernel1(){

__shared__ float conv_www[100];
//__shared__ float pad[28*28];
if(threadIdx.x<10&&threadIdx.y<10)
conv_www[threadIdx.x*10 + threadIdx.y]=conv_ww[threadIdx.x*10 + threadIdx.y+blockIdx.y*104];
;
//pad[threadIdx.x*10 + threadIdx.y]=padded_img[(threadIdx.x+4)*10 + (threadIdx.y+4)];

__syncthreads();

float sum = 0.0f;
      for(int i = 0; i < 10; ++i) {
      for(int j = 0; j <10 ; ++j) {
           if( threadIdx.x + j >= 4 && threadIdx.x + j < 33 && threadIdx.y + j >= 4 && threadIdx.y + j < 33) {
      sum += conv_www[i*10+j]* padded_img[(threadIdx.x + i)*33+ threadIdx.y + j];
           } 
      }  
      }
      
      
      
   if(sum+conv_b[blockIdx.y]>=0){
   reshape[blockIdx.y+threadIdx.x*28*8+threadIdx.y*8]=sum+conv_b[blockIdx.y];
   }
     else reshape[blockIdx.y+threadIdx.x*28*8+threadIdx.y*8]=0;
  
    
}//last 