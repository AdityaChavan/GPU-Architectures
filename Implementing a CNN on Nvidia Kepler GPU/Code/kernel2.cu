#include "hip/hip_runtime.h"
__global__ void kernel2(float*fc1_w ){

int h=threadIdx.x*28+threadIdx.y;
__shared__ float reshape2[28*28];
 reshape2[threadIdx.x*28+threadIdx.y]=reshape[blockIdx.y*28*28+h];
 __syncthreads();
atomicAdd(&I[blockIdx.x],reshape2[h]*fc1_w[(blockIdx.y*28*28+h)*512+blockIdx.x]);
}